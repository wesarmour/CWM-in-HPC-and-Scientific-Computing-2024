// In this assignment you will write a kernel for decimation of an integer 
// array by factor of two (which sums two neighbouring integer numbers into
// one). More description is in notes for this practical.

// Your tasks are:
// 1) to calculate decimation by factor of two using GPU and store 
//    the result to the host.
// 2) to calculate decimation by factor of four using GPU and store 
//    the result to the host.
// To do that you have to:
// 1) write the host code to manage memory, data initialization and data 
//    transfers
// 2) write GPU kernel which is appropriate for the tasks
// 3) configure grid for your GPU kernel

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

int Compare_results(int *GPU_output, int *h_input, size_t size, int DIT_factor){
	int itemp;
	size_t half_size;
	int error = 0;
	int nErrors = 0;
	int cislo = 0;
	
	half_size = size/DIT_factor;
	for(size_t f=0; f<half_size; f++){
		itemp=0;
		for(int t=0; t<DIT_factor; t++){
			itemp = itemp + h_input[f*DIT_factor + t];
		}
		error = (GPU_output[f]-itemp);
		if(error!=0 && cislo<20) {
			printf("f: %d; error: %d; GPU: %d; CPU: %d\n", f, error, GPU_output[f], itemp);
			cislo++;
		}
		if(error!=0) nErrors++;
	}
	
	return(nErrors);
}

void Check_errors(int *GPU_sum2, int *GPU_sum4, int *h_input, size_t size){
	int nErrors = 0;
	printf("Checking result for sum of two elements...\n");
	nErrors = Compare_results(GPU_sum2, h_input, size, 2);
	if(nErrors>0) printf("Correct solution and your solution does not match!\n");
	else printf("Your solution is correct!\n");
	
	printf("Checking result for sum of four elements...\n");
	nErrors = Compare_results(GPU_sum4, h_input, size, 4);
	if(nErrors>0) printf("Correct solution and your solution does not match!\n");
	else printf("Your solution is correct!\n");	
}

//----------------------------------------------------------------------
// TASK: Write your decimation GPU kernel
// When writing your kernel think carefully what each thread needs to read
// and what is it writes out and to where.
// Also consider what other threads might be reading and writing and where.

// put your kernel here

//----------------------------------------------------------------------



int main(void) {
	// Size of the array
	size_t N = 67108864;
	// Declaration of arrays used in this practical
	int *h_GPU_sum2, *h_GPU_sum4, *h_input;
	
	// memory allocation
	h_GPU_sum2 = (int*) malloc(N*sizeof(*h_GPU_sum2));
	h_GPU_sum4 = (int*) malloc(N*sizeof(*h_GPU_sum4));
	h_input       = (int*) malloc(N*sizeof(*h_input));
	if(h_GPU_sum2==NULL || h_GPU_sum4==NULL || h_input==NULL) return(1);
	
	// initiate host data
	for(size_t f=0; f<N; f++) {
		h_input[f] = f;
	}
	
	//----------------------------------------------------------------------
	// TASK: Write host code
	// Do not forget to initialize the device.
	// Remember that configuration of your grid depends on how you have 
	// written your decimation kernel
	
	// finish the host code here
	
	
	//----------------------------------------------------------------------
	
	Check_errors(h_GPU_sum2, h_GPU_sum4, h_input, N);
	
	// free memory on the host and the device
	free(h_GPU_sum2);
	free(h_GPU_sum4);
	free(h_input);
	
	hipDeviceReset();
	return (0);
}
